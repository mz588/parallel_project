#include "hip/hip_runtime.h"
// nvcc image_compression_parallel.cu -o parallel -lm -lcufft -w
// ./parallel 1

#include<stdint.h>
#include<stdlib.h>
#include<stdio.h>
#include<hipfft/hipfft.h>
#include<math.h>
#include<thrust/device_vector.h>
#include<thrust/copy.h>
#include<thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/complex.h>
#include <thrust/extrema.h>

// Necessary libs for reading in and writing to image files
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// #define STB_IMAGE_WRITE_IMPLEMENTATION
#define INPUTFILE "./input_images/image.png"
#define OUTPUT_GRAY_PNG "./result_images/image_gray.png"
#define OUTPUTFILE_JPG "./result_images/result_gray.jpg"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__host__ __device__ bool operator> (const hipfftDoubleComplex& a, const hipfftDoubleComplex& b){
  return sqrt(a.x * a.x + a.y * a.y) > sqrt(b.x * b.x + b.y * b.y);
}

__host__ __device__ bool operator< (const hipfftDoubleComplex& a, const hipfftDoubleComplex& b){
  return sqrt(a.x * a.x + a.y * a.y) < sqrt(b.x * b.x + b.y * b.y);
}


void printMatrix(double* m, int row, int col){
  for(int i = 0; i < row; i++){
    printf("[");
    for(int j = 0; j < col; j++){
      printf("%E", m[i*row+j]);
      if(j != col-1) printf(", ");
    }
    printf("]\n");
  }
  printf("\n");
}

void printImageData(unsigned char *m, int row, int col, int channels){
  for(int i = 0; i < row; i++){
    printf("[");
    for(int j = 0; j < col; j++){
      printf("(");
      for(int k = 0; k < channels; k++){
        printf("%d", (int)m[i*col*channels+j*channels+k]);
        if(k != channels - 1) printf(", ");
      }
      printf(")");
      if(j != col - 1) printf(", ");
    }
    printf("]\n");
  }
  printf("\n");
}

void printMatrixChars(unsigned char*m, int row, int col){
  for(int i = 0; i < row; i++){
    printf("[");
    for(int j = 0; j < col; j++){
      printf("%d", (int)m[IDX2C(i,j,row)]);
      if(j != col - 1) printf(", ");
    }
    printf("]\n");
  }
  printf("\n");
}

void printComplexMatrix(hipfftDoubleComplex *m, int row, int col){
  for(int i = 0; i < row; i++) {
    printf("[");
    for(int j = 0; j < col; j++){
      printf("(%E, %E)", m[IDX2C(i,j,row)].x, m[IDX2C(i,j,row)].y);
      if(j != col - 1) printf(", ");
    }
    printf("]\n");
  }
  printf("\n");
}

__global__ void generateResultImage(hipfftDoubleComplex *IFFT, hipfftDoubleComplex max, unsigned char*result, int row, int col) {
  int row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int col_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(row_index < row && col_index < col) {
    int result_index = row_index * col + col_index;
    int IFFT_index = IDX2C(row_index, col_index, row);
    result[result_index] = (unsigned char)(IFFT[IFFT_index].x/max.x*255);
  }
}

__global__ void toGrayScaleImage(unsigned char*src, unsigned char*dest, int row, int col, int origin_channels, int new_channels){
  int row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int col_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(row_index < row && col_index < col) {
    int src_starting = (row_index*col + col_index) * origin_channels;
    int dest_starting = (row_index*col + col_index) * new_channels;
    unsigned char red, green, blue;
    red = src[src_starting];
    green = src[src_starting+1];
    blue = src[src_starting+2];
    dest[dest_starting] = red*0.3+blue*0.11+green*0.59; 
    if(origin_channels == 4) dest[dest_starting+1] = src[src_starting+3];
  }
}

__global__ void toGrayScaleData(unsigned char*src, double*dest, int row, int col, int origin_channels){
  int row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int col_index = blockIdx.x * blockDim.x + threadIdx.x;
  if(row_index < row && col_index < col) {
    int src_starting = (row_index*col + col_index) * origin_channels;
    int dest_starting = row_index*col + col_index;
    double red, green, blue;
    red = (double)src[src_starting];
    green = (double)src[src_starting+1];
    blue = (double)src[src_starting+2]; 
    dest[dest_starting] = (red*0.3+blue*0.11+green*0.59)/255; 
  }
}

__global__ void copyToComplex(double*src, hipfftDoubleComplex*dest, int row, int col){
  int row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int col_index = blockIdx.x * blockDim.x + threadIdx.x;
  if(row_index < row && col_index < col) {
    int index = row_index * col + col_index;
    int dest_index = IDX2C(row_index, col_index, row);
    dest[dest_index].x = src[index];
  }
}

__global__ void cutoff(hipfftDoubleComplex*src, hipfftDoubleComplex*sorted, int thresholdIdx, int row, int col){
  int row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int col_index = blockIdx.x * blockDim.x + threadIdx.x;
  if(row_index < row && col_index < col) {
    int index = row_index*col + col_index;
    double threshold_val = sqrt(sorted[thresholdIdx].x * sorted[thresholdIdx].x + sorted[thresholdIdx].y * sorted[thresholdIdx].y);
    double cur_val = sqrt(src[index].x * src[index].x + src[index].y * src[index].x);
    if(cur_val < threshold_val) src[index].x = src[index].y = 0.0;
  }
}

void generateArray(hipfftDoubleComplex*dest, int row, int col){
  int index = 0;
  for(int i = 0; i < row; i++){
    for(int j = 0; j < col; j++) dest[IDX2C(i,j,row)].x = index++;
  }
}

int main(int argc, char* argv[]) {

  assert(argc == 2);
  double THRESHOLD = atof(argv[1]);

  // Pointer to the memory of image on device
  hipfftDoubleComplex *gray_image_dev, *fft_result, *Ifft_result;
  unsigned char *rgb_image_dev, *gray_image_chars_dev, *gray_image_chars;
  unsigned char *final_result_dev, *final_result;
  double *gray_image_doubles_dev;
  float elapsed_time;
  // declare cufft handle, use in each cufft call
  hipfftHandle planZ2Z, planIZ2Z;
  // for checking if cufft fails
  hipfftResult cuError;
  // for checking memory allocation on device fails
  hipError_t cudaStat = hipSuccess;
  // for measuring the time
  hipEvent_t start,stop;

  int width, height, origin_channels;

  // read in the image file
  unsigned char *rgb_image_chars = stbi_load(INPUTFILE, &width, &height, &origin_channels, 0);
  assert(rgb_image_chars != NULL);
  printf("width: %d, height: %d, origin_channels: %d\n", width, height, origin_channels);

  // Allocate memory on host
  int gray_channels = origin_channels == 4?2:1;
  gray_image_chars = (unsigned char*)calloc(height*width*gray_channels, sizeof(unsigned char));
  final_result = (unsigned char*)calloc(height*width, sizeof(unsigned char));

  // Allocate memory on device
  cudaStat = hipMalloc((void**)&gray_image_doubles_dev, sizeof(double) * height * width);
  assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&gray_image_chars_dev, sizeof(unsigned char)*height*width*gray_channels);
  assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&rgb_image_dev, sizeof(unsigned char)*height*width*origin_channels);
  assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&gray_image_dev , sizeof(hipfftDoubleComplex)*width*height);
  assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&final_result_dev, sizeof(unsigned char)*width*height);
  assert(cudaStat == hipSuccess);

  // allocate memory for result
  fft_result = (hipfftDoubleComplex *)calloc(height*width, sizeof(hipfftDoubleComplex));
  Ifft_result = (hipfftDoubleComplex*)calloc(width*height, sizeof(hipfftDoubleComplex));

  // create plans
  cuError = hipfftPlan2d(&planZ2Z, width, height, HIPFFT_Z2Z);
  assert(cuError == HIPFFT_SUCCESS);
  cuError = hipfftPlan2d(&planIZ2Z, width, height, HIPFFT_Z2Z);
  assert(cuError == HIPFFT_SUCCESS);

  //define block and grid dimensions
	const dim3 dimGrid((int)ceil((width)/16), (int)ceil((height)/16));
	const dim3 dimBlock(16, 16);

  // create event
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // start the timer
  hipEventRecord(start);

  // copy rgb image to device
  cudaStat = hipMemcpy(rgb_image_dev, rgb_image_chars, sizeof(unsigned char)*height*width*origin_channels, hipMemcpyHostToDevice);
  assert(cudaStat == hipSuccess);
  
  // Generate gray scale image
  toGrayScaleImage<<<dimGrid,dimBlock>>>(rgb_image_dev, gray_image_chars_dev, height, width, origin_channels, gray_channels);
  cudaStat = hipMemcpy(gray_image_chars, gray_image_chars_dev, sizeof(unsigned char)*height*width*gray_channels, hipMemcpyDeviceToHost);
  assert(cudaStat == hipSuccess);
  // save the result as a comparison
  stbi_write_png(OUTPUT_GRAY_PNG, width, height, gray_channels, gray_image_chars, width*gray_channels);

  // convert to double data 
  toGrayScaleData<<<dimGrid, dimBlock>>>(rgb_image_dev, gray_image_doubles_dev, height, width, origin_channels);

  // convert to hipfftComplex matrix
  copyToComplex<<<dimGrid, dimBlock>>>(gray_image_doubles_dev, gray_image_dev, height, width);

  // // Perform FFT on image
  thrust::device_vector<hipfftDoubleComplex> fft_result_dev(height*width);
  hipfftDoubleComplex *_fft_result_dev = (hipfftDoubleComplex *)thrust::raw_pointer_cast(fft_result_dev.data());
  cuError = hipfftExecZ2Z(planZ2Z, gray_image_dev, _fft_result_dev, HIPFFT_FORWARD);
  assert(cuError == HIPFFT_SUCCESS);

  // sorting on the copy of fft_result_dev (fft_result_sorted_dev)
  thrust::device_vector<hipfftDoubleComplex> fft_result_sorted_dev(height*width);
  thrust::copy(fft_result_dev.begin(), fft_result_dev.end(), fft_result_sorted_dev.begin());
  hipfftDoubleComplex * _fft_result_sorted_dev = (hipfftDoubleComplex *)thrust::raw_pointer_cast(fft_result_sorted_dev.data());
  thrust::sort(fft_result_sorted_dev.begin(), fft_result_sorted_dev.end(), thrust::greater<hipfftDoubleComplex>());
  
  // preserve only the values that are larger than the threshold value
  int cutoffIndex = height*width*(THRESHOLD);
  cutoff<<<dimGrid, dimBlock>>>(_fft_result_dev, _fft_result_sorted_dev, cutoffIndex, height, width);

  // Do the IFFT on the fft_result_dev
  thrust::device_vector<hipfftDoubleComplex> Ifft_result_dev(height*width);
  hipfftDoubleComplex * _Ifft_result_dev = (hipfftDoubleComplex *)thrust::raw_pointer_cast(Ifft_result_dev.data());
  cuError = hipfftExecZ2Z(planIZ2Z, _fft_result_dev, _Ifft_result_dev, HIPFFT_BACKWARD);
  assert(cuError == HIPFFT_SUCCESS);
  // find the max IFFT value
  thrust::device_vector<hipfftDoubleComplex>::iterator IFFT_max_iter = thrust::max_element(Ifft_result_dev.begin(), Ifft_result_dev.end());

  // Scale every data with the max 
  hipfftDoubleComplex IFFT_max = *IFFT_max_iter;
  generateResultImage<<<dimGrid, dimBlock>>>(_Ifft_result_dev, IFFT_max, final_result_dev, height, width);
  cudaStat = hipMemcpy(final_result, final_result_dev, sizeof(unsigned char)*height*width, hipMemcpyDeviceToHost);
  assert(cudaStat == hipSuccess);

  // stop the timer and compute time consumption
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time,start,stop);

  // write to the image file
  stbi_write_jpg(OUTPUTFILE_JPG, width, height, 1, final_result, 100);
  printf("Finishes writing to %s\n", OUTPUTFILE_JPG);
  printf("Total time consumption: %f\n",elapsed_time);

  stbi_image_free(rgb_image_chars); 
  free(gray_image_chars); free(fft_result); free(Ifft_result); free(final_result);
  hipFree(gray_image_dev); hipFree(final_result_dev);
  hipFree(rgb_image_dev); hipFree(gray_image_chars_dev);
  hipFree(gray_image_doubles_dev);

  return 0;
}