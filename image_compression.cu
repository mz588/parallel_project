#include "hip/hip_runtime.h"
#include<stdint.h>
#include<stdlib.h>
#include<stdio.h>
#include<hipfft/hipfft.h>
#include<math.h>

// Necessary libs for reading in and writing to image files
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// #define STB_IMAGE_WRITE_IMPLEMENTATION
#define INPUTFILE "image.png"
#define OUTPUTFILE_JPG "result.jpg"
#define OUTPUTFILE_PNG "result.png"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void printMatrix(double* m, int row, int col){
  for(int i = 0; i < row; i++){
    printf("[");
    for(int j = 0; j < col; j++){
      printf("%E", m[i*row+j]);
      if(j != col-1) printf(", ");
    }
    printf("]\n");
  }
  printf("\n");
}

void printImageData(unsigned char *m, int row, int col, int channels){
  for(int i = 0; i < row; i++){
    printf("[");
    for(int j = 0; j < col; j++){
      printf("(");
      for(int k = 0; k < channels; k++){
        printf("%d", (int)m[i*col*channels+j*channels+k]);
        if(k != channels - 1) printf(", ");
      }
      printf(")");
      if(j != col - 1) printf(", ");
    }
    printf("]\n");
  }
  printf("\n");
}

void printMatrixChars(unsigned char*m, int row, int col){
  for(int i = 0; i < row; i++){
    printf("[");
    for(int j = 0; j < col; j++){
      printf("%d", (int)m[IDX2C(i,j,row)]);
      if(j != col - 1) printf(", ");
    }
    printf("]\n");
  }
  printf("\n");
}

void printComplexMatrix(hipfftDoubleComplex *m, int row, int col){
  for(int i = 0; i < row; i++) {
    printf("[");
    for(int j = 0; j < col; j++){
      printf("(%E, %E)", m[IDX2C(i,j,row)].x, m[IDX2C(i,j,row)].y);
      if(j != col - 1) printf(", ");
    }
    printf("]\n");
  }
  printf("\n");
}

void toColMajorChars(unsigned char*src, unsigned char*dest, int row, int col){
  for(int i = 0; i < row; i++){
    for(int j = 0; j < col; j++){
      dest[IDX2C(i,j,row)] = src[i*col+j];
      // printf("src[%d*%d+%d]: %d, dest[%d]: %d\n", i,col,j,src[i*col+j],IDX2C(i,j,row), dest[IDX2C(i,j,row)]);
    }
  }
}

void toRowMajorChars(unsigned char*src, unsigned char*dest, int row, int col){
  for(int i = 0; i < row; i++){
    for(int j = 0; j < col; j++){
      dest[i*col+j] = src[IDX2C(i,j,row)];
      // printf("src[%d*%d+%d]: %d, dest[%d]: %d\n", i,col,j,dest[i*col+j],IDX2C(i,j,row), src[IDX2C(i,j,row)]);
    }
  }
}

void toGrayScaleImageSerial(unsigned char*src, unsigned char*dest, int row, int col, int origin_channels, int newChannels){
  for(int i = 0; i< row; i++){
    for(int j = 0; j < col; j++) {
      unsigned char red, green, blue;
      red = src[i*col*origin_channels+j*origin_channels];
      green = src[i*col*origin_channels+j*origin_channels+1];
      blue = src[i*col*origin_channels+j*origin_channels+2];
      // gray image default to have only one channel (not RGB 3 channels)
      dest[i*col*newChannels+j*newChannels] = red*0.3+blue*0.11+green*0.59; 
      // alpha channel only for png files
      if(origin_channels == 4) dest[i*col*newChannels+j*newChannels + 1] = src[i*col*origin_channels+j*origin_channels+3];
    }
  }
}

__global__ void toGrayScaleImage(unsigned char*src, unsigned char*dest, int row, int col, int origin_channels, int new_channels){
  int row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int col_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(row_index < row && col_index < col) {
    int src_starting = (row_index*col + col_index) * origin_channels;
    int dest_starting = (row_index*col + col_index) * new_channels;
    unsigned char red, green, blue;
    red = src[src_starting];
    green = src[src_starting+1];
    blue = src[src_starting+2];
    dest[dest_starting] = red*0.3+blue*0.11+green*0.59; 
    if(origin_channels == 4) dest[dest_starting+1] = src[src_starting+3];
  }
}

// input: rgb image, double matrix (row-major), height, width, rgb_channels
void toGrayScaleData(unsigned char*src, double*dest, int row, int col, int origin_channels){
  for(int i = 0; i < row; i++){
    for(int j = 0; j < col; j++){
      double red, green, blue;
      red = (double)src[i*col*origin_channels+j*origin_channels];
      green = (double)src[i*col*origin_channels+j*origin_channels+1];
      blue = (double)src[i*col*origin_channels+j*origin_channels+2]; 
      dest[i*col+j] = (red*0.3+blue*0.11+green*0.59)/255; 
    }
  }
}

void copyToComplex(double*src, hipfftDoubleComplex*dest, int row, int col){
  for(int i = 0; i < row; i++){
    for(int j = 0; j < col; j++){
      dest[IDX2C(i,j,row)].x = src[i*col+j];
    }
  }
}

void generateArray(hipfftDoubleComplex*dest, int row, int col){
  int index = 0;
  for(int i = 0; i < row; i++){
    for(int j = 0; j < col; j++) dest[IDX2C(i,j,row)].x = index++;
  }
}

int main(int argc, char* argv[]) {

  // Pointer to the memory of image on device
  hipfftDoubleComplex *gray_image_dev, *fft_result_dev, *fft_result, *Ifft_result, *Ifft_result_dev;
  unsigned char *rgb_image_dev, *gray_image_chars_dev, *gray_image_chars;
  unsigned char *final_result, *rgb_image_chars_back;
  // declare cufft handle, use in each cufft call
  hipfftHandle planZ2Z, planIZ2Z;
  // for checking if cufft fails
  hipfftResult cuError;
  // for checking memory allocation on device fails
  hipError_t cudaStat = hipSuccess;
  // for measuring the time
  hipEvent_t start,stop;

  int width, height, origin_channels;

  // read in the image file
  unsigned char *rgb_image_chars = stbi_load(INPUTFILE, &width, &height, &origin_channels, 0);
  assert(rgb_image_chars != NULL);
  printf("width: %d, height: %d, origin_channels: %d\n", width, height, origin_channels);

  // Allocate memory on host
  int gray_channels = origin_channels == 4?2:1;
  gray_image_chars = (unsigned char*)calloc(height*width*gray_channels, sizeof(unsigned char));
  final_result = (unsigned char*)calloc(height*width, sizeof(unsigned char));
  rgb_image_chars_back = (unsigned char*)calloc(height*width*origin_channels, sizeof(unsigned char));

  // Allocate memory on device
  cudaStat = hipMalloc((void**)&gray_image_chars_dev, sizeof(unsigned char)*height*width*gray_channels);
  assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&rgb_image_dev, sizeof(unsigned char)*height*width*origin_channels);
  assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&gray_image_dev , sizeof(hipfftDoubleComplex)*width*height);
  assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&fft_result_dev, sizeof(hipfftDoubleComplex)*width*height);
  assert(cudaStat == hipSuccess);
  cudaStat = hipMalloc((void**)&Ifft_result_dev, sizeof(hipfftDoubleComplex)*width*height);
  assert(cudaStat == hipSuccess);

  //define block and grid dimensions
	const dim3 dimGrid((int)ceil((width)/16), (int)ceil((height)/16));
	const dim3 dimBlock(16, 16);

  // copy rgb image to device
  cudaStat = hipMemcpy(rgb_image_dev, rgb_image_chars, sizeof(unsigned char)*height*width*origin_channels, hipMemcpyHostToDevice);
  assert(cudaStat == hipSuccess);
  // Generate gray scale image
  toGrayScaleImage<<<dimGrid,dimBlock>>>(rgb_image_dev, gray_image_chars_dev, height, width, origin_channels, gray_channels);
  // copy the gray image to host
  cudaStat = hipMemcpy(gray_image_chars, gray_image_chars_dev, sizeof(unsigned char)*height*width*gray_channels, hipMemcpyDeviceToHost);
  assert(cudaStat == hipSuccess);

  // save the result
  stbi_write_png(OUTPUTFILE_PNG, width, height, gray_channels, gray_image_chars, width*gray_channels);

  // // convert to double data
  // double *gray_image_doubles = (double*)calloc(height*width, sizeof(double));
  // toGrayScaleData(rgb_image_chars, gray_image_doubles, height, width, origin_channels);

  // // copy the image data to float and convert to col-major
  // hipfftDoubleComplex *gray_image_complex = (hipfftDoubleComplex*) calloc(height*width, sizeof(hipfftDoubleComplex));
  // copyToComplex(gray_image_doubles, gray_image_complex, height, width);

  // // allocate memory for result
  // fft_result = (hipfftDoubleComplex *)calloc(height*width, sizeof(hipfftDoubleComplex));
  // Ifft_result = (hipfftDoubleComplex*)calloc(width*height, sizeof(hipfftDoubleComplex));

  // // // copy the image data to device
  // cudaStat = hipMemcpy(gray_image_dev, gray_image_complex, sizeof(hipfftDoubleComplex)*height*width, hipMemcpyHostToDevice);
  // assert(cudaStat == hipSuccess);

  // // create plans
  // cuError = hipfftPlan2d(&planZ2Z, width, height, HIPFFT_Z2Z);
  // assert(cuError == HIPFFT_SUCCESS);
  // cuError = hipfftPlan2d(&planIZ2Z, width, height, HIPFFT_Z2Z);
  // assert(cuError == HIPFFT_SUCCESS);

  // // Perform FFT on image
  // cuError = hipfftExecZ2Z(planZ2Z, gray_image_dev, fft_result_dev, HIPFFT_FORWARD);
  // assert(cuError == HIPFFT_SUCCESS);

  // // Do the IFFT on the fft_result_dev
  // cuError = hipfftExecZ2Z(planIZ2Z,fft_result_dev, Ifft_result_dev, HIPFFT_BACKWARD);
  // assert(cuError == HIPFFT_SUCCESS);
  // cudaStat = hipMemcpy(Ifft_result, Ifft_result_dev, sizeof(hipfftDoubleComplex)*width*height, hipMemcpyDeviceToHost);
  // assert(cudaStat == hipSuccess);

  // // print result of inverse fft
  // // printComplexMatrix(Ifft_result, height, width);

  // // find the max value in the inverse fft result for scaling
  // // TODO: replace this with kernel function
  // double max = 0.0;
  // for(int i = 0; i < height; i++){
  //   for(int j = 0; j<width; j++){
  //     if(Ifft_result[i*width+j].x>max) max = Ifft_result[i*width+j].x;
  //   }
  // }

  // // Scale every data with the max 
  // for(int i = 0; i < height; i++){
  //   for(int j = 0; j < width; j++){
  //     final_result[i*width+j] = (unsigned char)((Ifft_result[IDX2C(i,j,height)].x)/max*255);
  //   }
  // }

  // // write to the image file
  // stbi_write_jpg(OUTPUTFILE_JPG, width, height, 1, final_result, 100);
  // printf("Finishes writing to %s\n", OUTPUTFILE_JPG);

  // stbi_image_free(rgb_image_chars); 
  // free(gray_image_chars); free(gray_image_doubles);
  // free(fft_result); free(gray_image_complex);
  // free(Ifft_result); free(final_result);
  // hipFree(fft_result_dev); hipFree(gray_image_dev);
  // hipFree(Ifft_result_dev);
  return 0;
}